
#include "cudpp/cudpp.h"
//#include <algorithm>
#include <stdio.h>

typedef unsigned int uint;

extern "C"
void checkCudaError(const char *msg)
{
#if defined(_DEBUG) || defined(DEBUG)
    hipError_t e = hipDeviceSynchronize();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error %s : %s\n", msg, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
    e = hipGetLastError();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error %s : %s\n", msg, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
#endif
}

